#include "hip/hip_runtime.h"
#include <glm/glm.hpp>
#include "particlefield.h"
#include "particle.h"
#include "vpmUtils.h"

__device__ void PedrizzettiRelaxation::relax(int index, ParticleField* field) {
    Particle& particle = field->particles[index];

    glm::vec3 omega = nablaCrossX(particle.J);

    particle.Gamma = (1.0f - relaxFactor) * particle.Gamma
                    + relaxFactor * glm::length(particle.Gamma) / glm::length(omega) * omega;
}

__device__ void CorrectedPedrizzettiRelaxation::relax(int index, ParticleField* field) {
    Particle& particle = field->particles[index];

    glm::vec3 omega = nablaCrossX(particle.J);

    float omegaNorm = glm::length(omega);
    float gammaNorm = glm::length(particle.Gamma);

    particle.Gamma = (1.0f - relaxFactor) * particle.Gamma
                    + relaxFactor * gammaNorm / omegaNorm * omega;

    particle.Gamma /= sqrt(1.0f - 2.0f * (1.0f - relaxFactor) * relaxFactor 
                        * (1.0f - glm::dot(particle.Gamma, omega) / (omegaNorm * gammaNorm)));
}