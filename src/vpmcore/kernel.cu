#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <string>
#include <random>
#include <memory>
#include "kernel.h"
#include "../lean_vtk.hpp"
#include "../vortexringsimulation.hpp"

__constant__ vpmfloat rungeKuttaCoefs[3][2] = {
    {0.0f, 1.0f / 3.0f},
    {-5.0f / 9.0f, 15.0f / 16.0f},
    {-153.0f / 128.0f, 8.0f / 15.0f}
};

// Constructor
Particle::Particle() 
    : X(0.0f), Gamma(0.0f), sigma(0.0f), vol(0.0f), circulation(0.0f), isStatic(false),
      U(0.0f), J(0.0f), PSE(0.0f), M(0.0f), C(0.0f), SFS(0.0f) {}

__host__ __device__ void Particle::reset() {
    U = vpmvec3(0.0f);
    J = vpmmat3(0.0f);
    PSE = vpmvec3(0.0f);
}

__host__ __device__ void Particle::resetSFS() {
    SFS = vpmvec3(0.0f);
}

template <typename R, typename S, typename K>
__global__ void PedrizzettiRelaxation::operator()(int N, ParticleField<R, S, K>* field) {
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) return;

    const vpmvec3 omega = nablaCrossX(field->particles[index].J);
    const vpmvec3 oldGamma = field->particles[index].Gamma;

    field->particles[index].Gamma; = (1.0 - relaxFactor) * oldGamma
                     + relaxFactor * glm::length(oldGamma) / glm::length(omega) * omega;
}

template <typename R, typename S, typename K>
 __global__ void CorrectedPedrizzettiRelaxation::operator()(int N, ParticleField<R, S, K>* field) {
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) return;

    const vpmvec3  omega = nablaCrossX(field->particles[index].J);
    const vpmvec3  oldGamma = field->particles[index].Gamma;
    const vpmfloat omegaNorm = glm::length(omega);
    const vpmfloat gammaNorm = glm::length(oldGamma);

    const vpmfloat tmp = sqrt(1.0 - 2.0 * (1.0 - relaxFactor) * relaxFactor
        * (1.0 - glm::dot(oldGamma, omega) / (omegaNorm * gammaNorm)));

    field->particles[index].Gamma = ((1.0 - relaxFactor) * oldGamma
        + relaxFactor * gammaNorm / omegaNorm * omega) / tmp;
}

template <typename R, typename S, typename K>
__global__ void NoRelaxation::operator()(int N, ParticleField<R, S, K>* field) {}

template <typename R, typename S, typename K>
__device__ void dynamicProcedure(int index, ParticleField<R, S, K>* field, vpmfloat alpha, vpmfloat relaxFactor,
                                 bool forcePositive, vpmfloat minC, vpmfloat maxC) {
#ifdef SHARED_MEMORY
    Particle& particle = s_particleBuffer[threadIdx.x];
#else
    Particle& particle = field->particles[index];
#endif

    const vpmfloat zeta0 = field->kernel.zeta(0);

    // CALCULATIONS WITH TEST FILTER
    field->particles[index].sigma *= alpha;
    __syncthreads();

    particle.reset();
    calcVelJacNaive(index, field);
    __syncthreads();

    particle.resetSFS();
    calcEstrNaive(index, field);

    // Clear temporary variable (not necessary?)
    // particle.M = vpmmat3{ 0.0f };

    // temporary variables
    particle.M[0] = xDotNablaY(particle.Gamma, particle.J);
    particle.M[1] = particle.SFS;

    // CALCULATIONS WITH DOMAIN FILTER
    field->particles[index].sigma /= alpha;
    __syncthreads();

    particle.reset();
    calcVelJacNaive(index, field);
    __syncthreads();

    particle.resetSFS();
    calcEstrNaive(index, field);

    // Save temporary variables
    particle.M[0] -= xDotNablaY(particle.Gamma, particle.J);
    particle.M[1] -= particle.SFS;

    // CALCULATE COEFFICIENT
    vpmfloat numerator = glm::dot(particle.M[0], particle.Gamma);
    numerator *= 3.0f * alpha - 2.0f;

    vpmfloat denominator = glm::dot(particle.M[1], particle.Gamma);
    denominator *= particle.sigma * particle.sigma * particle.sigma / zeta0;

    // Don't initialize denominator to 0
    if (particle.C[2] == 0) particle.C[2] = denominator;

    // Lagrangian average
    numerator   = relaxFactor * numerator   + (1 - relaxFactor) * particle.C[1];
    denominator = relaxFactor * denominator + (1 - relaxFactor) * particle.C[2];

    // Enforce maximum and minimum absolute values
    if (fabs(numerator/denominator) > maxC) {
        if (fabs(denominator) < fabs(particle.C[2])) denominator = copysign(particle.C[2], denominator);

        if (fabs(numerator/denominator) > maxC) numerator = copysign(denominator, numerator) * maxC;
    }
    else if (fabs(numerator/denominator) < minC) numerator = copysign(denominator, numerator) * minC;

    // Save numerator and denominator of model coefficient
    particle.C[1] = numerator;
    particle.C[2] = denominator;

    // Store model coefficient
    particle.C[0] = particle.C[1] / particle.C[2];

    // Force the coefficient to be positive
    if (forcePositive) particle.C[0] = fabs(particle.C[0]);

    // Clear temporary variable (not necessary?)
    // particle.M = vpmmat3{ 0.0f };
}

template <typename R, typename S, typename K>
__device__ void DynamicSFS::operator()(int index, ParticleField<R, S, K>* field, vpmfloat a, vpmfloat b, int numBlocks, int blockSize) {
    if (a == 1.0f || a == 0.0f) {
        dynamicProcedure(index, field, alpha, relaxFactor, forcePositive, minC, maxC);

        if (particle.C[0] * glm::dot(particle.Gamma, particle.SFS) < 0) particle.C[0] = 0;
    }
    else {
        calcVelJacNaive<<<numBlocks, blockSize>>>(field, field, field->kernel, true);
        calcEstrNaive<<<numBlocks, blockSize>>>(field, field, field->kernel, true);
    }
}

template <typename R, typename S, typename K>
void NoSFS::operator()(ParticleField<R, S, K>* field, vpmfloat a, vpmfloat b, int numBlocks, int blockSize) {
    resetParticlesSFS<<<numBlocks, blockSize>>>(field);
    calcVelJacNaive<<<numBlocks, blockSize >>>(field, field, field->kernel, true);
}

template <typename R, typename S, typename K>
__global__ void resetParticles(ParticleField<R, S, K>* field) {
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) return;

    field->particles[index].U = vpmvec3{ 0.0 };
    field->particles[index].J = vpmmat3{ 0.0 };
    field->particles[index].PSE = vpmvec3{ 0.0 };
}

template <typename R, typename S, typename K>
__global__ void resetParticlesSFS(ParticleField<R, S, K>* field) {
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) return;

    field->particles[index].SFS = vpmvec3{ 0.0 };
}


template <typename Rs, typename Ss, typename Ks, typename Rt, typename St, typename Kt, typename K>
__global__ void calcEstrNaive(ParticleField<Rs, Ss, Ks>* source, ParticleField<Rt, St, Kt>* target, K kernel) {
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) return;

    // Get required variables from global memory
    const vpmvec3 targetX = target->particles[index].X;
    const vpmmat3 targetJ = target->particles[index].J;
    vpmvec3 targetSFS = target->particles[index].SFS;

    for (int i = 0; i < source->np; ++i) {
        Particle& sourceParticle = source->particles[i];
        const vpmfloat sourceSigma = sourceParticle.sigma;

        targetSFS += kernel.zeta(glm::length(targetX - sourceParticle.X) / sourceSigma)
            / (sourceSigma * sourceSigma * sourceSigma)
            * xDotNablaY(sourceParticle.Gamma, targetJ - sourceParticle.J);
    }

    // Copy variables back to global memory
    target->particles[index].SFS = targetSFS;
}

template <typename Rs, typename Ss, typename Ks, typename Rt, typename St, typename Kt, typename K>
__global__ void calcVelJacNaive(ParticleField<Rs, Ss, Ks>* source, ParticleField<Rt, St, Kt>* target, K kernel, bool reset=false) {
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) return;

    // Get required variables from global memory
    const vpmvec3 targetX = target->particles[index].X;
    if (reset) {
        vpmvec3 targetU = vpmvec3{ 0.0 };
        vpmmat3 targetJ = vpmmat3{ 0.0 };
    }
    else {
        vpmvec3 targetU = target->particles[index].U;
        vpmmat3 targetJ = target->particles[index].J;
    }
    

    for (int i = 0; i < source->np; ++i) {
        if (i == index) continue;

        Particle& sourceParticle = source->particles[i];
        const vpmfloat invSourceSigma = 1 / sourceParticle.sigma;
        const vpmvec3 sourceGamma = sourceParticle.Gamma;

        const vpmvec3 dX = targetX - sourceParticle.X;
        const vpmfloat r = glm::length(dX);

        // is this needed?
        if (r < EPS) continue;

        // Kernel evaluation
        const vpmfloat g_sgm = kernel.g(r * invSourceSigma);
        const vpmfloat dg_sgmdr = kernel.dgdr(r * invSourceSigma);

        // Compute velocity
        const vpmvec3 crossProd = glm::cross(dX, sourceGamma) * (-const4 / (r*r*r));
        targetU += g_sgm * crossProd;

        // Compute Jacobian
        vpmfloat tmp = dg_sgmdr * invSourceSigma / r - 3.0 * g_sgm / (r*r);
        const vpmvec3 dX_norm = dX / r;

        for (int l = 0; l < 3; ++l) {
            for (int k = 0; k < 3; ++k) {
                targetJ[l][k] += tmp * crossProd[k] * dX_norm[l];
            }
        }

        tmp = - const4 * g_sgm / (r*r*r);

        // Account for kronecker delta term
        targetJ[0][1] -= tmp * sourceGamma[2];
        targetJ[0][2] += tmp * sourceGamma[1];
        targetJ[1][0] += tmp * sourceGamma[2];
        targetJ[1][2] -= tmp * sourceGamma[0];
        targetJ[2][0] -= tmp * sourceGamma[1];
        targetJ[2][1] += tmp * sourceGamma[0];
    }

    // Copy variables back to global memory
    target->particles[index].U = targetU;
    target->particles[index].J = targetJ;
}

template <typename R, typename S, typename K>
__global__ void rungekutta(int N, ParticleField<R, S, K>* field, vpmfloat dt, bool useRelax) {
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) return;

#ifdef SHARED_MEMORY
    s_particleBuffer[threadIdx.x] = field->particles[index];
    Particle& particle = s_particleBuffer[threadIdx.x];
#else
    Particle& particle = field->particles[index];
#endif

    vpmvec3 S;
    vpmfloat Z;
    vpmfloat zeta0 = field->kernel.zeta(0.0f);
    vpmvec3 Uinf = field->Uinf;
    R relax = field->relaxation;

    // Reset temp variable
    particle.M = vpmmat3{ 0.0f };

    // Loop over the pairs
    for (int i = 0; i < 3; ++i) {
        vpmfloat a = rungeKuttaCoefs[i][0];
        vpmfloat b = rungeKuttaCoefs[i][1];

        // RUN SFS
        field->SFS(index, field, a, b);
        __syncthreads();

        particle.M[0] = a * particle.M[0] + dt * (particle.U + Uinf);
        particle.X += b * particle.M[0];

        S = xDotNablaY(particle.Gamma, particle.J);
        Z = 0.2f * glm::dot(S, particle.Gamma) / glm::dot(particle.Gamma, particle.Gamma);

        particle.M[1] = a * particle.M[1] + dt * (S - 3 * Z * particle.Gamma -
            particle.C[0] * particle.SFS * particle.sigma * particle.sigma * particle.sigma / zeta0);
        particle.M[2][1] = a * particle.M[2][1] - dt * (particle.sigma * Z);

        particle.Gamma += b * particle.M[1];
        particle.sigma += b * particle.M[2][1];

#ifdef SHARED_MEMORY
        // Copy variables back to global memory
        field->particles[index].Gamma = particle.Gamma;
        field->particles[index].sigma = particle.sigma;
        field->particles[index].X     = particle.X;
#endif
        __syncthreads();
    }

    if (useRelax) {
        particle.reset();
        calcVelJacNaive(index, field);

        __syncthreads(); // useRelax is the same for all threads

        relax(particle);

#ifdef SHARED_MEMORY
        // Copy variables back to global memory
        field->particles[index].Gamma = particle.Gamma;
#endif
    }
}

void writeVTK(int numParticles, Particle* particleBuffer, std::string filename, int timestep) {
    const int dim = 3;

    leanvtk::VTUWriter writer;

    std::vector<double> particleX;
    std::vector<double> particleU;
    std::vector<double> particleGamma;
    std::vector<double> particleOmega;
    std::vector<double> particleSigma;
    std::vector<double> particleIdx;
    particleX.reserve(dim * numParticles);
    particleU.reserve(dim * numParticles);
    particleGamma.reserve(dim * numParticles);
    particleOmega.reserve(dim * numParticles);
    particleSigma.reserve(numParticles);
    particleIdx.reserve(numParticles);

    vpmvec3 omega;

    for (int i = 0; i < numParticles; ++i) {
        Particle& particle = particleBuffer[i];

        particleIdx.push_back(i);
        particleSigma.push_back(particle.sigma);

        omega = nablaCrossX(particle.J);

        for (int j = 0; j < dim; ++j) {
            particleU.push_back(particle.U[j]);
            particleX.push_back(particle.X[j]);
            particleGamma.push_back(particle.Gamma[j]);
            particleOmega.push_back(omega[j]);
        }
    }

    writer.add_scalar_field("index", particleIdx);
    writer.add_scalar_field("sigma", particleSigma);
    writer.add_vector_field("position", particleX, dim);
    writer.add_vector_field("velocity", particleU, dim);
    writer.add_vector_field("circulation", particleGamma, dim);
    writer.add_vector_field("vorticity", particleOmega, dim);
    writer.write_point_cloud("../output/" + filename + "_" + std::to_string(timestep) + ".vtu", dim, particleX);
}

template <typename R, typename S, typename K>
void runVPM(
    int maxParticles,
    int numParticles,
    int numTimeSteps,
    vpmfloat dt,
    int fileSaveSteps,
    vpmvec3 uInf,
    Particle* particleBuffer,
    R relaxation,
    S sfs,
    K kernel,
    std::string filename) {
    int fullBlocksPerGrid{ (numParticles + BLOCK_SIZE - 1) / BLOCK_SIZE };

    // Declare device particle buffer
    Particle* dev_particleBuffer;
    hipMalloc((void**)&dev_particleBuffer, maxParticles * sizeof(Particle));

    // Copy particle buffer from host to device
    hipMemcpy(dev_particleBuffer, particleBuffer, numParticles * sizeof(Particle), hipMemcpyHostToDevice);

    ParticleField<R, S, K> field{
        maxParticles,
        dev_particleBuffer,
        numParticles,
        0,
        0.0f,
        kernel,
        uInf,
        sfs,
        true,
        relaxation
    };

    // Declare device particle field and copy host field to device
    ParticleField<R, S, K>* dev_field;
    hipMalloc((void**)&dev_field, sizeof(ParticleField<R, S, K>));
    hipMemcpy(dev_field, &field, sizeof(ParticleField<R, S, K>), hipMemcpyHostToDevice);

    std::cout << particleBuffer[0].U.x << std::endl;

    writeVTK(numParticles, particleBuffer, filename, 0);

    for (int i = 1; i <= numTimeSteps; ++i) {
        rungekutta<R, S, K><<<fullBlocksPerGrid, blockSize>>>(
            numParticles, dev_field, dt, true
        );

        if (i % fileSaveSteps == 0) {
            hipMemcpy(particleBuffer, dev_particleBuffer, numParticles * sizeof(Particle), hipMemcpyDeviceToHost);
            //writeVTK(numParticles, particleBuffer, filename, i / fileSaveSteps);
            std::cout << particleBuffer[0].U.x << std::endl;
        }
    }

    // free device memory
    hipFree(dev_particleBuffer);
    hipFree(dev_field);
}

void randomCubeInit(Particle* particleBuffer, int N, vpmfloat cubeSize, vpmfloat maxCirculation, vpmfloat maxSigma) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<vpmfloat> uniform(-1, 1);
    std::uniform_real_distribution<vpmfloat> uniformPos(0, 1);

    for (int i = 0; i < N; ++i) {
        Particle& particle = particleBuffer[i];

        particle.sigma = maxSigma * uniformPos(gen);
        particle.Gamma = maxCirculation * uniform(gen) * glm::normalize(vpmvec3{ uniform(gen), uniform(gen), uniform(gen) });
        particle.circulation = glm::length(particle.Gamma);

        particle.X = cubeSize * uniform(gen) * glm::normalize(vpmvec3{ uniform(gen), uniform(gen), uniform(gen) });
    }
}

void randomSphereInit(Particle* particleBuffer, int N, vpmfloat sphereRadius, vpmfloat maxCirculation, vpmfloat maxSigma) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<vpmfloat> uniform(-1, 1);
    std::uniform_real_distribution<vpmfloat> uniformPos(0, 1);

    for (int i = 0; i < N; ++i) {
        Particle& particle = particleBuffer[i];

        particle.sigma = maxSigma * uniformPos(gen);
        particle.Gamma = maxCirculation * uniform(gen) * glm::normalize(vpmvec3{ uniform(gen), uniform(gen), uniform(gen) });
        particle.circulation = glm::length(particle.Gamma);

        vpmfloat theta = 2 * PI * uniformPos(gen);
        vpmfloat phi = PI * uniformPos(gen);
        vpmfloat radius = std::cbrt(uniformPos(gen)) * sphereRadius;

        vpmfloat x = radius * sin(phi) * cos(theta);
        vpmfloat y = radius * sin(phi) * sin(theta);
        vpmfloat z = radius * cos(phi);
        particle.X = vpmvec3{ x, y, z };
    }
}

void runSimulation() {
    // Define basic parameters
    int maxParticles{ 6000 };
    int numTimeSteps{ 10 };
    vpmfloat dt{ 0.01f };
    int numStepsVTK{ 1 };
    vpmvec3 uInf{ 0, 0, 0 };

    // Create host particle buffer
    Particle* particleBuffer = new Particle[maxParticles];
    // Initialize particle buffer
    //randomSphereInit(particleBuffer, maxParticles, 10.0f, 1.0f, 0.5f);
    int numParticles = initVortexRings(particleBuffer, maxParticles);

    // Run VPM method
    runVPM(
        maxParticles,
        numParticles,
        numTimeSteps,
        dt,
        numStepsVTK,
        uInf,
        particleBuffer,
        CorrectedPedrizzettiRelaxation(0.3f),
        NoSFS(),
        GaussianErfKernel(),
        "test"
    );

    // Free host particle buffer
    delete[] particleBuffer;
}