#include "hip/hip_runtime.h"
#include <glm/glm.hpp>
#include "particle.h"
#include "vpmUtils.h"
#include "velocities.h"
#include "subFilterScale.h"

template <typename Rs, typename Ss, typename Ks, typename Rt, typename St, typename Kt, typename K>
__device__ void calcEstrNaive(int index, ParticleField<Rs, Ss, Ks>* source, ParticleField<Rt, St, Kt>* target, K kernel) {
    Particle& targetParticle = target->particles[index];

    for (int i = 0; i < source->np; ++i) {
        Particle& sourceParticle = source->particles[i];

        glm::vec3 S = xDotNablaY(sourceParticle.Gamma, targetParticle.J - sourceParticle.J);

        glm::vec3 dX = targetParticle.X - sourceParticle.X;
        float r = glm::length(dX);

        targetParticle.SFS += kernel.zeta(r / sourceParticle.sigma) / powf(sourceParticle.sigma, 3.0f) * S;
    }
}

template <typename R, typename S, typename K>
__device__ void calcEstrNaive(int index, ParticleField<R, S, K>* field) {
    calcEstrNaive(index, field, field, field->kernel);
}

template <typename R, typename S, typename K>
__device__ void dynamicProcedure(int index, ParticleField<R, S, K>* field, float alpha, float relaxFactor,
                                 bool forcePositive, float minC, float maxC) {
    Particle& particle = field->particles[index];

    // CALCULATIONS WITH TEST FILTER
    particle.sigma *= alpha;

    particle.reset();
    calcVelJacNaive(index, field);

    particle.resetSFS();
    calcEstrNaive(index, field);

    // Clear temporary variable (really necessary?)
    particle.M = glm::mat3{ 0.0f };

    // temporary variables
    particle.M[0] = xDotNablaY(particle.Gamma, particle.J);
    particle.M[1] = particle.SFS;

    // CALCULATIONS WITH DOMAIN FILTER
    particle.sigma /= alpha;

    particle.reset();
    calcVelJacNaive(index, field);

    particle.resetSFS();
    calcEstrNaive(index, field);

    // Save temporary variables
    particle.M[0] -= xDotNablaY(particle.Gamma, particle.J);
    particle.M[1] -= particle.SFS;

    // CALCULATE COEFFICIENT
    float numerator = glm::dot(particle.M[0], particle.Gamma);
    numerator *= 3.0f * alpha - 2.0f;

    float denominator = glm::dot(particle.M[1], particle.Gamma);
    denominator *= particle.sigma * particle.sigma * particle.sigma / field->kernel.zeta(0);

    // Don't initialize denominator to 0
    if (particle.C[2] == 0) particle.C[2] = denominator;

    // Lagrangian average
    numerator   = relaxFactor * numerator   + (1 - relaxFactor) * particle.C[1];
    denominator = relaxFactor * denominator + (1 - relaxFactor) * particle.C[2];

    // Enforce maximum and minimum absolute values
    if (fabs(numerator/denominator) > maxC) {
        if (fabs(denominator) < fabs(particle.C[2])) denominator = copysign(particle.C[2], denominator);

        if (fabs(numerator/denominator) > maxC) numerator = copysign(denominator, numerator) * maxC;
    }
    else if (fabs(numerator/denominator) < minC) numerator = copysign(denominator, numerator) * minC;

    // Save numerator and denominator of model coefficient
    particle.C[1] = numerator;
    particle.C[2] = denominator;

    // Store model coefficient
    particle.C[0] = particle.C[1] / particle.C[2];

    // Force the coefficient to be positive
    if (forcePositive) particle.C[0] = fabs(particle.C[0]);

    // Clear temporary variable
    particle.M = glm::mat3{ 0.0f };
}

template <typename R, typename S, typename K>
__device__ void DynamicSFS::operator()(int index, ParticleField<R, S, K>* field, float a, float b) {
    Particle& particle = field->particles[index];

    if (a == 1.0f || a == 0.0f) {
        dynamicProcedure(index, field, alpha, relaxFactor, forcePositive, minC, maxC);

        if (particle.C[0] * glm::dot(particle.Gamma, particle.SFS) < 0) particle.C[0] = 0;
    }
    else {
        particle.reset();
        calcVelJacNaive(index, field);

        particle.resetSFS();
        calcEstrNaive(index, field);
    }
}


template <typename R, typename S, typename K>
__device__ void NoSFS::operator()(int index, ParticleField<R, S, K>* field, float a, float b) {
    Particle& particle = field->particles[index];

    particle.reset();

    calcVelJacNaive(index, field);
}